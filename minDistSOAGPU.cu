#include "hip/hip_runtime.h"
/****
Author: David Tran
File: minDistSOAGPU.cu
Compilation: nvcc minDistSOAGPU.cu -Wno-deprecated-gpu-targets -o  minDistSOAGPU
Execution: dtran7.csc656@tiger:~$ ./minDistSOAGPU
***/

#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// You may edit NUMPARTICLES and THREADSPERBLOCK for measurements
#define NUMPARTICLES 32768
#define THREADSPERBLOCK 4

void initPos(float *);
void findMinsG(float *pos, int *minIndex, float *minDistance);
void dumpResults(int index[], float d[]);
__global__ void findMinsGPU(float *p, int *minI, float *minD);
__device__ float findDistanceGPU(float *p, int i, int j);
// You are not allowed to change main()!
int main() {
  hipEvent_t start, stop;
  float time;

  float *pos;
  int *minIndex;
  float *minDistance;

  pos = (float *) malloc(NUMPARTICLES * 3 * sizeof(float));
  minIndex = (int *) malloc(NUMPARTICLES * sizeof(int));
  minDistance = (float *) malloc(NUMPARTICLES * sizeof(float));

  initPos(pos);

  // create timer events
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start, 0);

  findMinsG(pos, minIndex, minDistance);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);

  printf("%d particles, %d threads per block\n", NUMPARTICLES, THREADSPERBLOCK);
  printf("Elapsed time = %f\n", time);

  dumpResults(minIndex, minDistance);

}

void initPos(float *p) {
  // this should be identical to initPos() for minDistSOA.c
  // your code goes here
  for(int i=0; i<NUMPARTICLES; i++){
    p[i] = rand() / (float) RAND_MAX;
    p[NUMPARTICLES+i] = rand() / (float) RAND_MAX;
    p[NUMPARTICLES*2+i] = rand() / (float) RAND_MAX;
   }
}

void findMinsG(float *pos, int *minIndex, float *minDistance) {
 // wrapper function for CUDA code
 // CUDA memory management and kernel calls go here
 float *dPos, *mDistance;
 int *mIndex;
 hipMalloc((void **) &dPos, NUMPARTICLES * 3 * sizeof(float));
 hipMalloc((void **) &mDistance, NUMPARTICLES * sizeof(float));
 hipMalloc((void **) &mIndex, NUMPARTICLES * sizeof(int));
 hipMemcpy(dPos, pos, NUMPARTICLES * 3 * sizeof(float), hipMemcpyHostToDevice);
 hipMemcpy(mDistance, minDistance, NUMPARTICLES * sizeof(float), hipMemcpyHostToDevice);
 hipMemcpy(mIndex, minIndex, NUMPARTICLES * sizeof(int), hipMemcpyHostToDevice);

 // Invoke kernal finMinsGPU()
 findMinsGPU<<<NUMPARTICLES/THREADSPERBLOCK, THREADSPERBLOCK>>>(dPos, mIndex, mDistance);
 hipDeviceSynchronize();
 hipMemcpy(minIndex, mIndex, NUMPARTICLES * sizeof(int), hipMemcpyDeviceToHost);
 hipMemcpy(minDistance, mDistance, NUMPARTICLES * sizeof(float), hipMemcpyDeviceToHost);
 /****** Did not want to change main so left this here to show that i remembered
 // clean up
 free(hm);
 hipFree(dm);
 free(hcs);
 hipFree(dcs);
 *****/
}
/* device function to find distances */
__device__ float findDistanceGPU(float *p, int i, int j) {
 float dx, dy, dz;

 dx = p[i] - p[j];
 dy = p[NUMPARTICLES + i] - p[NUMPARTICLES + j];
 dz = p[NUMPARTICLES*2 + i] - p[NUMPARTICLES*2 + j];

 return(dx*dx + dy*dy + dz*dz);
}

/* kernal function that calculates min distance */
__global__ void findMinsGPU(float *p, int *minI, float *minD) {
 // your kernel code goes here
  int i, j;
  float distance, mD;
  int mI;
  mD = 0;
  i = blockDim.x * blockIdx.x + threadIdx.x;
  if(i!=0){
  mI = 0;
     mD = findDistanceGPU(p, i, 0);
  }else{
     mI=1;
     mD = findDistanceGPU(p, 0, 1);
  }

  for(j=0; j<NUMPARTICLES; j++){
     if(i!=j) {
     // calculate distance between particles i and j
         distance = findDistanceGPU(p, i, j);
     // if distance < mD
         if(distance < mD){
            mD = distance;
            mI = j;
          }
      }
   }
minI[i] = mI;
minD[i] = mD;
}

void dumpResults(int index[], float d[]) {
int i;
FILE *fp;

fp = fopen("./dump.out", "w");

for (i=0; i<NUMPARTICLES; i++) {
fprintf(fp, "%d %d %f\n", i, index[i], d[i]);
}

fclose(fp);
}
